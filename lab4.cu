#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "lab4.cuh"

#define USE_SIMPLE_FILTER 0

//Round a / b to nearest higher integer value
int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

/*
    Transpose a 2D array (see SDK transpose example)
*/
extern "C"
void transpose(uint *d_src, uint *d_dest, uint width, int height)
{
    dim3 grid(iDivUp(width, BLOCK_DIM), iDivUp(height, BLOCK_DIM), 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
    d_transpose<<< grid, threads >>>(d_dest, d_src, width, height);
    getLastCudaError("Kernel execution failed");
}

// 8-bit RGBA version
extern "C"
void gaussianFilterRGBA(uint *d_src, uint *d_dest, uint *d_temp, int width, int height, float sigma, int order, int nthreads)
{
    // compute filter coefficients
    const float
    nsigma = sigma < 0.1f ? 0.1f : sigma,
    alpha = 1.695f / nsigma,
    ema = (float)std::exp(-alpha),
    ema2 = (float)std::exp(-2*alpha),
    b1 = -2*ema,
    b2 = ema2;

    float a0 = 0, a1 = 0, a2 = 0, a3 = 0, coefp = 0, coefn = 0;  
   
    const float k = (1-ema)*(1-ema)/(1+2*alpha*ema-ema2);
    a0 = k;
    a1 = k*(alpha-1)*ema;
    a2 = k*(alpha+1)*ema;
    a3 = -k*ema2;      
    coefp = (a0+a1)/(1+b1+b2);
    coefn = (a2+a3)/(1+b1+b2);

    // process columns
#if USE_SIMPLE_FILTER
    d_simpleRecursive_rgba<<< iDivUp(width, nthreads), nthreads >>>(d_src, d_temp, width, height, ema);
#else
    d_recursiveGaussian_rgba<<< iDivUp(width, nthreads), nthreads >>>(d_src, d_temp, width, height, a0, a1, a2, a3, b1, b2, coefp, coefn);
#endif
    getLastCudaError("Kernel execution failed");

    transpose(d_temp, d_dest, width, height);
    getLastCudaError("transpose: Kernel execution failed");

    // process rows

    d_recursiveGaussian_rgba<<< iDivUp(height, nthreads), nthreads >>>(d_dest, d_temp, height, width, a0, a1, a2, a3, b1, b2, coefp, coefn);

    getLastCudaError("Kernel execution failed");

    transpose(d_temp, d_dest, height, width);
}
